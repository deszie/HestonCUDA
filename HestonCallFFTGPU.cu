#include "hip/hip_runtime.h"
#include "HestonCallFFTGPU.hpp"
#include "HestonCUDA.hpp"
#include <complex>
#define _USE_MATH_DEFINES
#include <cmath>
#include <gsl/gsl_spline.h>
#include <iostream>

// NVIDIA CUDA Headers
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

// NVIDIA Thrust Headers (http://developer.nvidia.com/Thrust)
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

// NVIDIA CUFFT
#include <hipfft/hipfft.h>

__host__ __device__ static __inline__ hipDoubleComplex simpsonWIndex(int index) {
  index &= 3;  
  switch (index) {
    case 0:
      return make_hipDoubleComplex(0.0, -1.0);
    case 1:
      return make_hipDoubleComplex(-1.0, 0.0);
    case 2:
      return make_hipDoubleComplex(0.0, 1.0);
    case 3:
      return make_hipDoubleComplex(1.0, 0.0);
  }
  return make_hipDoubleComplex(0.0, 0.0);
}

struct HestonCallFFTGPU_functor {
  double dKappa;
  double dTheta;
  double dSigma;
  double dRho;
  double dV0;
  double dR;
  double dT;
  double dS0;
  double dStrike;

  double dX0;
  double dAlpha;
  double dEta;
  double dB;

  HestonCallFFTGPU_functor(
    double dKappa,   // rate of reversion
    double dTheta,   // int run variance
    double dSigma,   // vol of vol
    double dRho,     // correlation
    double dV0,      // initial variance
    double dR,       // instantaneous short rate
    double dT,       // time till maturity
    double dS0,      // initial asset price
    double dStrike,

    double dX0,
    double dAlpha,
    double dEta,
    double dB
  ) : dKappa(dKappa), dTheta(dTheta), dSigma(dSigma), dRho(dRho), dV0(dV0), dR(dR), dT(dT), dS0(dS0), dStrike(dStrike), dX0(dX0), dAlpha(dAlpha), dEta(dEta), dB(dB) {}

  __host__ __device__
  hipDoubleComplex operator() (int index) {
    hipDoubleComplex zI      = make_hipDoubleComplex(0.0, 1.0);
    double dU               = index * dEta;

    hipDoubleComplex zV      = make_hipDoubleComplex(dU, -(dAlpha + 1.0));
    hipDoubleComplex zZeta   = mul(-0.5, hipCadd(hipCmul(zV, zV), hipCmul(zI, zV)));
    hipDoubleComplex zGamma  = sub(dKappa, mul(dRho * dSigma, hipCmul(zV, zI)));
    hipDoubleComplex zPHI    = sqrt(hipCsub(hipCmul(zGamma, zGamma), mul(2.0 * dSigma * dSigma, zZeta)));
    
    hipDoubleComplex zA      = mul(dX0 + dR * dT, hipCmul(zI, zV));
    hipDoubleComplex zB      = mul(dV0, hipCdiv(mul(2.0, hipCmul(zZeta, sub(1.0, exp(mul(-dT, zPHI))))), hipCsub(mul(2.0, zPHI), hipCmul(hipCsub(zPHI, zGamma), sub(1.0, exp(mul(-dT, zPHI)))))));
    hipDoubleComplex zC      = mul(-dKappa * dTheta / (dSigma * dSigma), hipCadd(mul(2.0, log(hipCdiv(hipCsub(mul(2.0, zPHI), hipCmul(hipCsub(zPHI, zGamma), sub(1.0, exp(mul(-dT, zPHI))))), (mul(2.0, zPHI))))), mul(dT, hipCsub(zPHI, zGamma))));


    hipDoubleComplex zCharFunc = exp(hipCadd(hipCadd(zA, zB), zC));
    hipDoubleComplex zModifiedCharFunc = hipCdiv(mul(exp(-dR * dT), zCharFunc), add(dAlpha * dAlpha + dAlpha - dU * dU, make_hipDoubleComplex(0.0, dU * (2.0 * dAlpha + 1.0))));

    hipDoubleComplex zSimpsonW = mul(1.0 / 3.0, add(3.0, simpsonWIndex(index)));
    if (index == 0) zSimpsonW.x -= 1.0 / 3.0;

    return mul(dEta, hipCmul(hipCmul(exp(make_hipDoubleComplex(0.0, dB * dU)), zModifiedCharFunc), zSimpsonW));
  }
};

double HestonCallFFTGPU(
  double dKappa,   // rate of reversion
  double dTheta,   // int run variance
  double dSigma,   // vol of vol
  double dRho,     // correlation
  double dV0,      // initial variance
  double dR,       // instantaneous short rate
  double dT,       // time till maturity
  double dS0,      // initial asset price
  double dStrike,
  long   lN) {
  std::complex<double> zI(0.0, 1.0);

  double dX0 = log(dS0);
  double dAlpha = 1.5;
  // double dC = 600;
  double dEta = 0.25;
  double dB = M_PI / dEta;

  std::complex<double> zFFTFunc[lN];
  std::complex<double> zPayoff[lN];
  double               dPayoff[lN];

  double dLambda = 2 * dB / lN;
  double dPosition = (log(dStrike) + dB) / dLambda + 1;

  thrust::device_vector<int> dev_zFFTFuncI(lN);
  thrust::device_vector<hipDoubleComplex> dev_zFFTFunc(lN);
  
  thrust::sequence(dev_zFFTFuncI.begin(), dev_zFFTFuncI.end());
  thrust::transform(dev_zFFTFuncI.begin(), dev_zFFTFuncI.end(), dev_zFFTFunc.begin(), HestonCallFFTGPU_functor(dKappa, dTheta, dSigma, dRho, dV0, dR, dT, dS0, dStrike, dX0, dAlpha, dEta, dB));

  thrust::copy(dev_zFFTFunc.begin(), dev_zFFTFunc.end(), (hipDoubleComplex*)zFFTFunc);

  hipfftHandle p;
  hipfftDoubleComplex* cufftFFTFunc = NULL;
  hipfftDoubleComplex* cufftPayoff  = NULL;

  hipMalloc((void**)&cufftFFTFunc, sizeof(hipfftDoubleComplex) * lN);
  hipMalloc((void**)&cufftPayoff, sizeof(hipfftDoubleComplex) * lN);

  hipMemcpy(cufftFFTFunc, zFFTFunc, sizeof(hipfftDoubleComplex) * lN, hipMemcpyHostToDevice);

  hipfftPlan1d(&p, lN, HIPFFT_Z2Z, 1);
  hipfftExecZ2Z(p, cufftFFTFunc, cufftPayoff, HIPFFT_FORWARD);
  
  hipMemcpy(zPayoff, cufftPayoff, sizeof(hipfftDoubleComplex) * lN, hipMemcpyDeviceToHost);

  hipfftDestroy(p);
  hipFree(cufftFFTFunc);
  hipFree(cufftPayoff);


  for (int i = 0; i < lN; i++) dPayoff[i] = zPayoff[i].real();

  double dCallValueM[lN];

  /* wchan: replace this later w/ the appropriate BLAS vector-scalar function */
  for (int i = 0; i < lN; i++) dCallValueM[i] = dPayoff[i] / M_PI;

  double dLin[lN];
  for (int i = 0; i < lN; i++) dLin[i] = 1.0 + i;

  gsl_interp_accel* acc = gsl_interp_accel_alloc();
  gsl_spline* spline = gsl_spline_alloc(gsl_interp_cspline, lN);
  gsl_spline_init(spline, dLin, dCallValueM, lN);

  double dPrice = exp(-log(dStrike) * dAlpha) * gsl_spline_eval(spline, dPosition, acc);

  gsl_spline_free(spline);
  gsl_interp_accel_free(acc);

  return dPrice;
}

