#include "hip/hip_runtime.h"
#include "HestonCallFFTGPU.hpp"
#include <complex>
#define _USE_MATH_DEFINES
#include <cmath>
#include <fftw3.h>
#include <gsl/gsl_spline.h>
#include <iostream>

// NVIDIA CUDA Headers
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

// NVIDIA Thrust Headers (http://developer.nvidia.com/Thrust)
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

// NVIDIA CUFFT
#include <hipfft/hipfft.h>

__host__ __device__ static __inline__ double mag(hipDoubleComplex c) {
  return sqrt(c.x * c.x + c.y * c.y);
}

__host__ __device__ static __inline__ double phase(hipDoubleComplex c) {
  return atan(c.y / c.x);
}

__host__ __device__ static __inline__ hipDoubleComplex mul(double s, hipDoubleComplex c) {
  return make_hipDoubleComplex(s * c.x, s * c.y);
}

__host__ __device__ static __inline__ hipDoubleComplex sub(double s, hipDoubleComplex c) {
  return make_hipDoubleComplex(s - c.x, c.y);
}

__host__ __device__ static __inline__ hipDoubleComplex add(double s, hipDoubleComplex c) {
  return make_hipDoubleComplex(s + c.x, c.y);
}

__host__ __device__ static __inline__ hipDoubleComplex sqrt(hipDoubleComplex c) {
  double f = mag(c);
  double hp = 0.5 * phase(c);
  
  return make_hipDoubleComplex(f * cos(hp), f * sin(hp));
}

__host__ __device__ static __inline__ hipDoubleComplex exp(hipDoubleComplex c) {
  double f = exp(c.x);

  return make_hipDoubleComplex(f * cos(c.y), f * sin(c.y));
}

__host__ __device__ static __inline__ hipDoubleComplex log(hipDoubleComplex c) {
  return make_hipDoubleComplex(log(mag(c)), phase(c));
}

__host__ __device__ static __inline__ hipDoubleComplex simpsonWIndex(int index) {
  index &= 3;  
  switch (index) {
    case 0:
      return make_hipDoubleComplex(0.0, -1.0);
    case 1:
      return make_hipDoubleComplex(-1.0, 0.0);
    case 2:
      return make_hipDoubleComplex(0.0, 1.0);
    case 3:
      return make_hipDoubleComplex(1.0, 0.0);
  }
  return make_hipDoubleComplex(0.0, 0.0);
}

struct HestonCallFFTGPU_functor {
  double dKappa;
  double dTheta;
  double dSigma;
  double dRho;
  double dV0;
  double dR;
  double dT;
  double dS0;
  double dStrike;

  double dX0;
  double dAlpha;
  double dEta;
  double dB;

  HestonCallFFTGPU_functor(
    double dKappa,   // rate of reversion
    double dTheta,   // int run variance
    double dSigma,   // vol of vol
    double dRho,     // correlation
    double dV0,      // initial variance
    double dR,       // instantaneous short rate
    double dT,       // time till maturity
    double dS0,      // initial asset price
    double dStrike,

    double dX0,
    double dAlpha,
    double dEta,
    double dB
  ) : dKappa(dKappa), dTheta(dTheta), dSigma(dSigma), dRho(dRho), dV0(dV0), dR(dR), dT(dT), dS0(dS0), dStrike(dStrike), dX0(dX0), dAlpha(dAlpha), dEta(dEta), dB(dB) {}

  __host__ __device__
  hipDoubleComplex operator() (int index) {
    hipDoubleComplex zI      = make_hipDoubleComplex(0.0, 1.0);

    double dU               = index * dEta;
    hipDoubleComplex zV      = make_hipDoubleComplex(dU, dAlpha + 1.0);
    hipDoubleComplex zZeta   = mul(0.5, hipCadd(hipCmul(zV, zV), hipCmul(zI, zV)));
    hipDoubleComplex zGamma  = sub(dKappa, mul(dRho * dSigma, hipCmul(zV, zI)));
    hipDoubleComplex zPHI    = sqrt(hipCsub(hipCmul(zGamma, zGamma), mul(2.0 * dSigma * dSigma, zZeta)));
    hipDoubleComplex zA      = mul(dX0 + dR * dT, hipCmul(zI, zV));
    hipDoubleComplex zB      = mul(dV0, hipCdiv(mul(2.0, hipCmul(zZeta, sub(1, exp(mul(-dT, zPHI))))), hipCsub(mul(2.0, zPHI), hipCmul(hipCsub(zPHI, zGamma), sub(1.0, exp(mul(-dT, zPHI)))))));
    hipDoubleComplex zC      = mul(-dKappa * dTheta / (dSigma * dSigma), hipCadd(mul(2.0, log(hipCdiv(hipCsub(mul(2.0, zPHI), hipCmul(hipCsub(zPHI, zGamma), sub(1.0, exp(mul(-dT, zPHI))))), (mul(2.0, zPHI))))), mul(dT, hipCsub(zPHI, zGamma))));


    hipDoubleComplex zCharFunc = exp(hipCadd(hipCadd(zA, zB), zC));
    hipDoubleComplex zModifiedCharFunc = hipCdiv(mul(exp(-dR * dT), zCharFunc), add(dAlpha * dAlpha + dAlpha - dU * dU, make_hipDoubleComplex(0.0, dU * (2.0 * dAlpha + 1.0))));

    hipDoubleComplex zSimpsonW = mul(1.0 / 3.0, add(3.0, simpsonWIndex(index)));
    if (index == 0) zSimpsonW.x -= 1.0 / 3.0;

    return mul(dEta, hipCmul(hipCmul(exp(make_hipDoubleComplex(0.0, dB * dU)), zModifiedCharFunc), zSimpsonW));
  }
};

double HestonCallFFTGPU(
  double dKappa,   // rate of reversion
  double dTheta,   // int run variance
  double dSigma,   // vol of vol
  double dRho,     // correlation
  double dV0,      // initial variance
  double dR,       // instantaneous short rate
  double dT,       // time till maturity
  double dS0,      // initial asset price
  double dStrike,
  long   lN) {
  std::complex<double> zI(0.0, 1.0);

  double dX0 = log(dS0);
  double dAlpha = 1.5;
  // double dC = 600;
  double dEta = 0.25;
  double dB = M_PI / dEta;

  std::complex<double> zFFTFunc[lN];
  std::complex<double> zPayoff[lN];
  double               dPayoff[lN];

  double dLambda = 2 * dB / lN;
  double dPosition = (log(dStrike) + dB) / dLambda + 1;
  
  thrust::device_vector<int> dev_zFFTFuncI(lN);
  thrust::device_vector<hipDoubleComplex> dev_zFFTFunc(lN);
  
  thrust::sequence(dev_zFFTFuncI.begin(), dev_zFFTFuncI.end());
  thrust::transform(dev_zFFTFuncI.begin(), dev_zFFTFuncI.end(), dev_zFFTFunc.begin(), HestonCallFFTGPU_functor(dKappa, dTheta, dSigma, dRho, dV0, dR, dT, dS0, dStrike, dX0, dAlpha, dEta, dB));

  thrust::copy(dev_zFFTFunc.begin(), dev_zFFTFunc.end(), (hipDoubleComplex*)zFFTFunc);

  fftw_complex* fftwFFTFunc = reinterpret_cast<fftw_complex*>(zFFTFunc);
  fftw_complex* fftwPayoff  = reinterpret_cast<fftw_complex*>(zPayoff);

  fftw_plan p = fftw_plan_dft_1d(lN, fftwFFTFunc, fftwPayoff, FFTW_FORWARD, FFTW_ESTIMATE);
  fftw_execute(p);
  fftw_destroy_plan(p);
  
  /*
  fftw_complex* fftwFFTFunc = reinterpret_cast<fftw_complex*>(zFFTFunc);
  fftw_complex* fftwPayoff  = reinterpret_cast<fftw_complex*>(zPayoff);

  fftw_plan p = fftw_plan_dft_1d(lN, fftwFFTFunc, fftwPayoff, FFTW_FORWARD, FFTW_ESTIMATE);
  fftw_execute(p);
  fftw_destroy_plan(p);
  */

  for (int i = 0; i < lN; i++) dPayoff[i] = zPayoff[i].real();

  double dCallValueM[lN];

  /* wchan: replace this later w/ the appropriate BLAS vector-scalar function */
  for (int i = 0; i < lN; i++) dCallValueM[i] = dPayoff[i] / M_PI;

  double dLin[lN];
  for (int i = 0; i < lN; i++) dLin[i] = 1.0 + i;

  gsl_interp_accel* acc = gsl_interp_accel_alloc();
  gsl_spline* spline = gsl_spline_alloc(gsl_interp_cspline, lN);
  gsl_spline_init(spline, dLin, dCallValueM, lN);

  double dPrice = exp(-log(dStrike) * dAlpha) * gsl_spline_eval(spline, dPosition, acc);

  gsl_spline_free(spline);
  gsl_interp_accel_free(acc);

  return dPrice;
}

