#include "hip/hip_runtime.h"
#include "HestonCallFFTGPU.hpp"
#include "HestonCUDA.hpp"
#include <complex>
#define _USE_MATH_DEFINES
#include <cmath>
#include <gsl/gsl_spline.h>
#include <iostream>

// NVIDIA CUDA Headers
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

// NVIDIA Thrust Headers (http://developer.nvidia.com/Thrust)
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

// NVIDIA CUFFT
#include <hipfft/hipfft.h>

__host__ __device__ static __inline__ hipDoubleComplex simpsonWIndex(int index) {
  index &= 3;  
  switch (index) {
    case 0:
      return make_hipDoubleComplex(0.0, -1.0);
    case 1:
      return make_hipDoubleComplex(-1.0, 0.0);
    case 2:
      return make_hipDoubleComplex(0.0, 1.0);
    case 3:
      return make_hipDoubleComplex(1.0, 0.0);
  }
  return make_hipDoubleComplex(0.0, 0.0);
}

struct HestonCallFFTGPU_functor {
  HestonCUDAPrecision dKappa;
  HestonCUDAPrecision dTheta;
  HestonCUDAPrecision dSigma;
  HestonCUDAPrecision dRho;
  HestonCUDAPrecision dV0;
  HestonCUDAPrecision dR;
  HestonCUDAPrecision dT;
  HestonCUDAPrecision dS0;
  HestonCUDAPrecision dStrike;

  HestonCUDAPrecision dX0;
  HestonCUDAPrecision dAlpha;
  HestonCUDAPrecision dEta;
  HestonCUDAPrecision dB;

  HestonCallFFTGPU_functor(
    HestonCUDAPrecision dKappa,   // rate of reversion
    HestonCUDAPrecision dTheta,   // int run variance
    HestonCUDAPrecision dSigma,   // vol of vol
    HestonCUDAPrecision dRho,     // correlation
    HestonCUDAPrecision dV0,      // initial variance
    HestonCUDAPrecision dR,       // instantaneous short rate
    HestonCUDAPrecision dT,       // time till maturity
    HestonCUDAPrecision dS0,      // initial asset price
    HestonCUDAPrecision dStrike,

    HestonCUDAPrecision dX0,
    HestonCUDAPrecision dAlpha,
    HestonCUDAPrecision dEta,
    HestonCUDAPrecision dB
  ) : dKappa(dKappa), dTheta(dTheta), dSigma(dSigma), dRho(dRho), dV0(dV0), dR(dR), dT(dT), dS0(dS0), dStrike(dStrike), dX0(dX0), dAlpha(dAlpha), dEta(dEta), dB(dB) {}

  __host__ __device__
  hipDoubleComplex operator() (int index) {
    hipDoubleComplex zI      = make_hipDoubleComplex(0.0, 1.0);
    HestonCUDAPrecision dU               = index * dEta;

    hipDoubleComplex zV      = make_hipDoubleComplex(dU, -(dAlpha + 1.0));
    hipDoubleComplex zZeta   = mul(-0.5, hipCadd(hipCmul(zV, zV), hipCmul(zI, zV)));
    hipDoubleComplex zGamma  = sub(dKappa, mul(dRho * dSigma, hipCmul(zV, zI)));
    hipDoubleComplex zPHI    = sqrt(hipCsub(hipCmul(zGamma, zGamma), mul(2.0 * dSigma * dSigma, zZeta)));
    
    hipDoubleComplex zA      = mul(dX0 + dR * dT, hipCmul(zI, zV));
    hipDoubleComplex zB      = mul(dV0, hipCdiv(mul(2.0, hipCmul(zZeta, sub(1.0, exp(mul(-dT, zPHI))))), hipCsub(mul(2.0, zPHI), hipCmul(hipCsub(zPHI, zGamma), sub(1.0, exp(mul(-dT, zPHI)))))));
    hipDoubleComplex zC      = mul(-dKappa * dTheta / (dSigma * dSigma), hipCadd(mul(2.0, log(hipCdiv(hipCsub(mul(2.0, zPHI), hipCmul(hipCsub(zPHI, zGamma), sub(1.0, exp(mul(-dT, zPHI))))), (mul(2.0, zPHI))))), mul(dT, hipCsub(zPHI, zGamma))));


    hipDoubleComplex zCharFunc = exp(hipCadd(hipCadd(zA, zB), zC));
    hipDoubleComplex zModifiedCharFunc = hipCdiv(mul(exp(-dR * dT), zCharFunc), add(dAlpha * dAlpha + dAlpha - dU * dU, make_hipDoubleComplex(0.0, dU * (2.0 * dAlpha + 1.0))));

    hipDoubleComplex zSimpsonW = mul(1.0 / 3.0, add(3.0, simpsonWIndex(index)));
    if (index == 0) zSimpsonW.x -= 1.0 / 3.0;

    return mul(dEta, hipCmul(hipCmul(exp(make_hipDoubleComplex(0.0, dB * dU)), zModifiedCharFunc), zSimpsonW));
  }
};

HestonCUDAPrecision HestonCallFFTGPU(
  HestonCUDAPrecision dKappa,   // rate of reversion
  HestonCUDAPrecision dTheta,   // int run variance
  HestonCUDAPrecision dSigma,   // vol of vol
  HestonCUDAPrecision dRho,     // correlation
  HestonCUDAPrecision dV0,      // initial variance
  HestonCUDAPrecision dR,       // instantaneous short rate
  HestonCUDAPrecision dT,       // time till maturity
  HestonCUDAPrecision dS0,      // initial asset price
  HestonCUDAPrecision dStrike,
  long   lN) {
  std::complex<HestonCUDAPrecision> zI(0.0, 1.0);

  HestonCUDAPrecision dX0 = log(dS0);
  HestonCUDAPrecision dAlpha = 1.5;
  // HestonCUDAPrecision dC = 600;
  HestonCUDAPrecision dEta = 0.25;
  HestonCUDAPrecision dB = M_PI / dEta;

  std::complex<HestonCUDAPrecision> zFFTFunc[lN];
  std::complex<HestonCUDAPrecision> zPayoff[lN];
  HestonCUDAPrecision               dPayoff[lN];

  HestonCUDAPrecision dLambda = 2 * dB / lN;
  HestonCUDAPrecision dPosition = (log(dStrike) + dB) / dLambda + 1;

  thrust::device_vector<int> dev_zFFTFuncI(lN);
  thrust::device_vector<HestonCUDAPrecisionComplex> dev_zFFTFunc(lN);
  
  thrust::sequence(dev_zFFTFuncI.begin(), dev_zFFTFuncI.end());
  thrust::transform(dev_zFFTFuncI.begin(), dev_zFFTFuncI.end(), dev_zFFTFunc.begin(), HestonCallFFTGPU_functor(dKappa, dTheta, dSigma, dRho, dV0, dR, dT, dS0, dStrike, dX0, dAlpha, dEta, dB));

  thrust::copy(dev_zFFTFunc.begin(), dev_zFFTFunc.end(), (hipDoubleComplex*)zFFTFunc);

  hipfftHandle p;
#if defined HestonCUDAPrecisionSingle
  cufftSingleComplex* cufftFFTFunc = NULL;
  cufftSingleComplex* cufftPayoff  = NULL;
#elif defined HestonCUDAPrecisionDouble
  hipfftDoubleComplex* cufftFFTFunc = NULL;
  hipfftDoubleComplex* cufftPayoff  = NULL;
#endif

  hipMalloc((void**)&cufftFFTFunc, sizeof(HestonCUDAPrecisionComplex) * lN);
  hipMalloc((void**)&cufftPayoff, sizeof(HestonCUDAPrecisionComplex) * lN);

  hipMemcpy(cufftFFTFunc, zFFTFunc, sizeof(hipfftDoubleComplex) * lN, hipMemcpyHostToDevice);

#if defined HestonCUDAPrecisionSingle
  hipfftPlan1d(&p, lN, HIPFFT_C2C, 1);
  hipfftExecC2C(p, cufftFFTFunc, cufftPayoff, HIPFFT_FORWARD);
#elif defined HestonCUDAPrecisionDouble
  hipfftPlan1d(&p, lN, HIPFFT_Z2Z, 1);
  hipfftExecZ2Z(p, cufftFFTFunc, cufftPayoff, HIPFFT_FORWARD);
#endif

  hipMemcpy(zPayoff, cufftPayoff, sizeof(HestonCUDAPrecisionComplex) * lN, hipMemcpyDeviceToHost);

  hipfftDestroy(p);
  hipFree(cufftFFTFunc);
  hipFree(cufftPayoff);


  for (int i = 0; i < lN; i++) dPayoff[i] = zPayoff[i].real();

  HestonCUDAPrecision dCallValueM[lN];

  /* wchan: replace this later w/ the appropriate BLAS vector-scalar function */
  for (int i = 0; i < lN; i++) dCallValueM[i] = dPayoff[i] / M_PI;

  HestonCUDAPrecision dLin[lN];
  for (int i = 0; i < lN; i++) dLin[i] = 1.0 + i;

  gsl_interp_accel* acc = gsl_interp_accel_alloc();
  gsl_spline* spline = gsl_spline_alloc(gsl_interp_cspline, lN);
  gsl_spline_init(spline, dLin, dCallValueM, lN);

  HestonCUDAPrecision dPrice = exp(-log(dStrike) * dAlpha) * gsl_spline_eval(spline, dPosition, acc);

  gsl_spline_free(spline);
  gsl_interp_accel_free(acc);

  return dPrice;
}

